
#include <hip/hip_runtime.h>
/***
  Implementation of Spatial Transformer Networks[1]

  Under Simplified BSD License
  by Che-Wei Lin

  [1] Max Jaderberg et al. Spatial Transformer Networks. NIPS 2015
***/

__global__ void AffineForward(const float* bottom_data,
    const int* bs, const float* affine, const int len, float* top_data) {
    // bs = bottomSize

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len) return;

    // get current index, [h,w,c,n]
    int h = index % bs[0];
    int w = (index / bs[0]) % bs[1];
    int c = (index / bs[0] / bs[1]) % bs[2];
    int n = index / bs[0] / bs[1] / bs[2];

    // get current affine start index
    const float* a = affine + n*6;

    // calc bottom index
    //                    [a0 a1 0]
    // [x y 1] = [u v 1] *[a2 a3 0] 
    //                    [a4 a5 1]
    float nw = 2.0*((float)w/(float)bs[1]-0.5); //-1~1
    float nh = 2.0*((float)h/(float)bs[0]-0.5); //-1~1

    float w_new = ((a[0]*nw + a[2]*nh + a[4])/2.0+0.5)*(float)bs[1];
    float h_new = ((a[1]*nw + a[3]*nh + a[5])/2.0+0.5)*(float)bs[0];

    // calc neighbor pixel index, if > size or < size, do
    float v = 0.0;
    for (int x = floor(w_new); x <= ceil(w_new); x++) {
      for (int y = floor(h_new); y <= ceil(h_new); y++) {
        if (x < 0 || x>= bs[1] || y < 0 || y >= bs[0]){
          v = 0.0;
        }else{
          v = bottom_data[n*len/bs[3] + c*bs[1]*bs[0] + x*bs[0] + y];
        }
        top_data[index] += v * (1-abs(w_new - (float)x)) * (1-abs(h_new - (float)y));
      }
    }

}

__global__ void AffineBackward(const float* bottom_data,
    const int* bs, const float* affine, const int len, const float* top_data, const float* top_diff, float* bottom_diff1, float* bottom_diff2) {
    // bs = bottomSize

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len) return;

    // get current index, [h,w,c,n]
    int h = index % bs[0];
    int w = (index / bs[0]) % bs[1];
    int c = (index / bs[0] / bs[1]) % bs[2];
    int n = index / bs[0] / bs[1] / bs[2];

    // get current affine start index
    const float* a = affine + n*6;

    // calc bottom index
    //                    [a0 a1 0]
    // [x y 1] = [u v 1] *[a2 a3 0] 
    //                    [a4 a5 1]
    float nw = 2.0*((float)w/(float)bs[1]-0.5); //-1~1
    float nh = 2.0*((float)h/(float)bs[0]-0.5); //-1~1

    float w_new = ((a[0]*nw + a[2]*nh + a[4])/2.0+0.5)*(float)bs[1];
    float h_new = ((a[1]*nw + a[3]*nh + a[5])/2.0+0.5)*(float)bs[0];

    float u = 0.0;
    float dx = 0.0;
    float dy = 0.0;
    for (int x = max(floor(w_new),0.0); x <= min(ceil(w_new),(float)bs[1]); x++) {
      for (int y = max(floor(h_new),0.0); y <= min(ceil(h_new),(float)bs[0]); y++) {
          u = bottom_data[n*bs[2]*bs[1]*bs[0] + c*bs[1]*bs[0] + x*bs[0] + y];
          atomicAdd(bottom_diff1 + (n*bs[2]*bs[1]*bs[0] + c*bs[1]*bs[0] + x*bs[0] + y),  top_diff[index] * (1-abs(w_new - (float)x)) * (1-abs(h_new - (float)y))  );
          dx += u * (1-abs(h_new - (float)y)) * ((float)x >= w_new ? 1.0:-1.0 );
          dy += u * (1-abs(w_new - (float)x)) * ((float)y >= h_new ? 1.0:-1.0 );
      }
    }
    
    atomicAdd((bottom_diff2+n*6)+0, nw *dx*top_diff[index]);
    atomicAdd((bottom_diff2+n*6)+2, nh *dx*top_diff[index]);
    atomicAdd((bottom_diff2+n*6)+4, 1.0*dx*top_diff[index]);
    atomicAdd((bottom_diff2+n*6)+1, nw *dy*top_diff[index]);
    atomicAdd((bottom_diff2+n*6)+3, nh *dy*top_diff[index]);
    atomicAdd((bottom_diff2+n*6)+5, 1.0*dy*top_diff[index]);
}